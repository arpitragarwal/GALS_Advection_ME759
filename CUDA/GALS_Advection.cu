//
//  GALS - Before compiling the program, update the section of this program that is in the beginning of main and update all the functions in Initializer.h
//
//  Created by Raunak Bardia on 10/22/14.
//
// DISCLAIMER:
// Use the indexes carefully
// First index of array represents movement along y-direction because it represents rows
// Second index of array represents movement along x-direction because it represents columns
//
// Implementing GALS for a given initial level set function
// in a specified velocity field for a grid of cells
//
// Given -
// 1. Defining function at t = 0 which implies that phi and psi values are available for all node points at t = 0
// 2. Given velocity for the complete domain at all times
//
// All required data is stored in separate 2D matrices of phi, psix, psiy and psixy
// Boundary Condition grad(velocity).n > 0
//

// THIS IMPLEMENTATION WON'T WORK IF THE GRID IS SMALLER THAN (2 X 2)
#include <iostream>
#include <iomanip>
#include <math.h>
#include <stdio.h>
#include <fstream>
#include <sys/time.h>
#include <time.h>
#include <string.h>
#include <vector>
#include <tuple>
#include <hip/hip_runtime.h>
#include "Allocation.h"
//#include "InitializeLevelSet.h"

//Including Kernel
#include "AdvectionPointCalcsCUDA.cu"
#include "VortexVelocityCUDA.cu"

// y direction is the first index of array, x direction is the second index of array

using namespace std;

int main(){

    /* UPDATE ALL THE FOLLOWING VALUES */
    double xlim1 = 0.0;                       //Lower limit on x-axis
    double xlim2 = 1.0;                      //Upper limit on x-axis
    unsigned int nx = 127 + 1;                         //Number of nodes in x-direction INCLUDING THE EXTREME VALUES

    double ylim1 = 0.0;                       //Lower limit on y-axis
    double ylim2 = 1.0;                     //Upper limit on y-axis
    unsigned int ny = 127 + 1;                        //Number of nodes INCLUDING THE EXTREME VALUES

    double dt = (1.0/128.0);                     //Length of time step
    double Tfinal = 1.0;                    //Total time period for the simulation
    unsigned int option = 1;                         //Option - if you need animation initialize at 1 else initialize at 2
    unsigned int printstep = 8;                      //How frequently do you want to store the images (every nth time step)
    char psischeme[] = "SuperConsistent";   //'SuperConsistent' or 'Heuns'
    char backtrace_scheme[] = "RK3" ;      //'Euler' or 'RK3'
    double T_period = 1.0;                  //Period of the velocity field
    unsigned int TileSize = 16;

    //---------------------------------------------------------------------------------------------------------
    //MAKE SURE THAT YOU HAVE ENOUGH MEMORY SPACE IF YOU ARE STORING A LOT OF TIME STEP VALUES BECAUSE IT STORES ACROSS GRID POINTS FOR EACH PRINTSTEP

    /* USER UPDATE OVER */
    unsigned long gridmemory = nx * ny * sizeof(double);
    unsigned long gridmemoryint = nx * ny * sizeof(unsigned int);
    unsigned int n = Tfinal/dt; //Number of time steps
    if(option != 1)
        printstep = n;

    // Node Locations
    double dx = (xlim2 - xlim1)/(nx - 1);
    double dy = (ylim2 - ylim1)/(ny - 1);
    double* x = (double*) malloc(nx * sizeof(double));
    double* y = (double*) malloc(ny * sizeof(double));
    gridnodes(x,y,xlim1,ylim1,dx,dy,nx,ny);
    double *devicex, *devicey;
    // allocate device memory for x and y
    hipMalloc((void**)&devicex,nx * sizeof(double));	// Allocating GPU memory for the x-node values
    hipMalloc((void**)&devicey,ny * sizeof(double));	// Allocating GPU memory for the y-node values
    // Copy data from host to GPU
    hipMemcpy(devicex, x, nx * sizeof(double), hipMemcpyHostToDevice);	// Writing to device memory
    hipMemcpy(devicey, y, ny * sizeof(double), hipMemcpyHostToDevice);	// Writing to device memory

    // level set matrices
    double* mphi = (double*) malloc(gridmemory);
    double* mpsix = (double*) malloc(gridmemory);
    double* mpsiy = (double*) malloc(gridmemory);
    double* mpsixy = (double*) malloc(gridmemory);
    // Initializing at t = 0
    allocate_levelset_matrices(mphi,mpsix,mpsiy,mpsixy,x,y,nx,ny); //Initializing level set matrices

    double *masterdphi, *masterdpsix, *masterdpsiy;

    // allocate device memory for integer grids
    hipMalloc((void**)&masterdphi,gridmemory);	// Allocating GPU memory for the x-node values
    hipMalloc((void**)&masterdpsix,gridmemory);	// Allocating GPU memory for the y-node values
    hipMalloc((void**)&masterdpsiy,gridmemory);	// Allocating GPU memory for the y-node values
    double *dphi, *dpsix, *dpsiy, *dpsixy;
    // allocate device memory for integer grids
    hipMalloc((void**)&dphi,gridmemory);	// Allocating GPU memory for the x-node values
    hipMalloc((void**)&dpsix,gridmemory);	// Allocating GPU memory for the y-node values
    hipMalloc((void**)&dpsiy,gridmemory);	// Allocating GPU memory for the y-node values
    hipMalloc((void**)&dpsixy,gridmemory);	// Allocating GPU memory for the y-node values
    hipMemcpy(masterdphi, mphi, gridmemory, hipMemcpyHostToDevice);	// Writing to device memory
    hipMemcpy(masterdpsix, mpsix, gridmemory, hipMemcpyHostToDevice);	// Writing to device memory
    hipMemcpy(masterdpsiy, mpsiy, gridmemory, hipMemcpyHostToDevice);	// Writing to device memory
    hipMemcpy(dphi, mphi, gridmemory, hipMemcpyHostToDevice);	// Writing to device memory
    hipMemcpy(dpsix, mpsix, gridmemory, hipMemcpyHostToDevice);	// Writing to device memory
    hipMemcpy(dpsiy, mpsiy, gridmemory, hipMemcpyHostToDevice);	// Writing to device memory
    hipMemcpy(dpsixy, mpsixy, gridmemory, hipMemcpyHostToDevice);	// Writing to device memory

    // Removing existing files with these names if any
    remove("phi.txt");
    remove("psix.txt");
    remove("psiy.txt");
    remove("psixy.txt");
    remove("details.txt");
    remove("Velocity_x.txt");
    remove("Velocity_y.txt");
    fileprint(mphi,mpsix,mpsiy,mpsixy,nx,ny,x,y,0.0,T_period);
    ofstream details;
    details.open("details.txt", ios::out | ios::app);
    details<< nx << "," << ny << "," << std::fixed << std::setprecision(10) << dx << "," << dy << "," << xlim1 << "," << xlim2 << "," << ylim1 << "," << ylim2 << "," << n << "," << dt << "," << printstep;
    details.close();

    ///*
    // TIME STEPPING LOOP
    // If only the initial and final profiles are needed
    for(unsigned int t = 0; t < n; t++){

    double *dxadv, *dyadv;
    // allocate device memory for x and y
    hipMalloc((void**)&dxadv,gridmemory);	// Allocating GPU memory for the x-node values
    hipMalloc((void**)&dyadv,gridmemory);	// Allocating GPU memory for the y-node values

    unsigned int *dcellx, *dcelly,*dtracker;
    // allocate device memory for integer grids
    hipMalloc((void**)&dcellx,gridmemoryint);	// Allocating GPU memory for the x-node values
    hipMalloc((void**)&dcelly,gridmemoryint);	// Allocating GPU memory for the y-node values
    hipMalloc((void**)&dtracker,gridmemoryint);	// Allocating GPU memory for the y-node values

    dim3 dimBlock(TileSize, TileSize);
    dim3 dimGrid(nx/dimBlock.x, ny/dimBlock.y);

    // Find the point from which advection occurs at this time step
    advection_point_cuda<<<dimGrid,dimBlock>>>(devicex,devicey,dxadv,dyadv,nx,t,dt,T_period,TileSize);

    // Find the cell in which those advection points lie
    find_advection_point_location_cuda<<<dimGrid,dimBlock>>>(devicex,devicey,dxadv,dyadv,nx,ny,dcellx,dcelly,dtracker,xlim1,xlim2,ylim1,ylim2,TileSize);

    // Update the level set values
    update_levelset_data_cuda<<<dimGrid,dimBlock>>>(devicex, devicey, dxadv, dyadv, dcellx, dcelly, dtracker, t, dt, dphi, dpsix, dpsiy, dpsixy, masterdphi, masterdpsix, masterdpsiy,psischeme,backtrace_scheme,T_period,nx,ny,TileSize);

    devicetodevicecopy<<<dimGrid,dimBlock>>>(dphi,dpsix,dpsiy,masterdphi,masterdpsix,masterdpsiy,nx,TileSize);

    // Update the mixed derivatives now for the remaining grid points
    update_mixed_derivatives<<<dimGrid,dimBlock>>>(dpsix, dpsiy, dpsixy, nx, ny, dx, dy,TileSize);

    hipDeviceSynchronize();

    //---------------------------------------------------------------------------------------------------------
    // Feeding phi, psix, psiy and psixy values in their respective files
    if((t+1) % printstep == 0)
    {
        hipMemcpy(mphi, masterdphi, gridmemory, hipMemcpyDeviceToHost);       // Writing back to host memory
        hipMemcpy(mpsix, masterdpsix, gridmemory, hipMemcpyDeviceToHost);       // Writing back to host memory
        hipMemcpy(mpsiy, masterdpsiy, gridmemoryint, hipMemcpyDeviceToHost);      // Writing back to host memory
        //hipMemcpy(mpsixy, masterdpsixy, gridmemoryint, hipMemcpyDeviceToHost);  // Writing back to host memory
        fileprint(mphi,mpsix,mpsiy,mpsixy,nx,ny,x,y,(t+1)*dt,T_period);
    }
    cout<< t+1;
    cout<< " Time Step Completed" <<'\n';

    //---------------------------------------------------------------------------------------------------------
    //xadv.clear();
    //yadv.clear();
    //tracker.clear();
    //cellx.clear();
    //celly.clear();
    hipFree(dxadv);
    hipFree(dyadv);
    hipFree(dcellx);
    hipFree(dcelly);
    hipFree(dtracker);
    }  // end of time marching loop
     //*/
    return 0;
}
