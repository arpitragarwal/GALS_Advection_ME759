#include "hip/hip_runtime.h"
//
//  SymmetricVelocityCUDA.cu
//
//
//  Created by Raunak Bardia on 12/18/17.
//
//

#ifndef _SymmetricVelocityCUDA_cu
#define _SymmetricVelocityCUDA_cu

__device__ __host__ double Velx(double x, double y, double t, double T_period){
    double temp;
    temp = 0.5 * pi * cos(pi * t/T_period) * ((2*sin(2*pi*xc)*sin(2*pi*yc))*(x-xc)+2*(-cos(2*pi*xc)*(sin(pi*yc)*sin(pi*yc))+(sin(pi*xc)*sin(pi*xc))*cos(2*pi*yc))*(y-yc) );
    return temp;
}

__device__ __host__ double Vely(double x, double y,double t, double T_period){
    double temp;
    temp = 0.5 * pi * cos(pi * t/T_period) * ((-2*sin(2*pi*xc)*sin(2*pi*yc))*(y-yc)+2*(-cos(2*pi*xc)*(sin(pi*yc)*sin(pi*yc)) +(sin(pi*xc)*sin(pi*xc))*cos(2*pi*yc))*(x-xc) );
    return temp;
}

__device__ double gradUx(double x, double y, double t, double T_period){
    double temp;
    temp = 0.5*pi*cos(pi*t/T_period)*( ( 2*sin(2*pi*xc)*sin(2*pi*yc)));
    return temp;
}

__device__ double gradUy(double x, double y, double t, double T_period){
    double temp;
    temp = 0.5*pi*cos(pi*t/T_period)*(2*(-cos(2*pi*xc)*pow(sin(pi*yc),2)+pow(sin(pi*xc),2)*cos(2*pi*yc)) );
    return temp;
}

__device__ double gradVx(double x, double y, double t, double T_period){
    double temp;
    temp = gradUy(x,y,t,T_period);
    return temp;
}

__device__ double gradVy(double x, double y, double t, double T_period){
    double temp;
    temp = -gradUx(x,y,t,T_period);
    return temp;
}
#endif
