#include "hip/hip_runtime.h"
//
//  OscillatingVelocityCUDA.cu
//
//
//  Created by Raunak Bardia on 12/18/17.
//
//

#ifndef _OscillatingVelocityCUDA_cu
#define _OscillatingVelocityCUDA_cu

__device__ __host__ double Velx(double x, double y, double t, double T_period){
    double temp;
    double signum = cos(pi * t/T_period);
    if(signum == 0)
        signum = 0;
    else if(signum < 0)
        signum = -1;
    else
        signum = 1;
    temp = (50.0 - x) * signum;
    return temp;
}

__device__ __host__ double Vely(double x, double y,double t, double T_period){
    double temp;
    double signum = cos(pi * t/T_period);
    if(signum == 0)
        signum = 0;
    else if(signum < 0)
        signum = -1;
    else
        signum = 1;
    temp = (y - 50.0) * signum;
    return temp;
}

__device__ double gradUx(double x, double y, double t, double T_period){
    double temp;
    double signum = cos(pi * t/T_period);
    if(signum == 0)
        signum = 0;
    else if(signum < 0)
        signum = -1;
    else
        signum = 1;
    temp = -1 * signum;
    return temp;
}

__device__ double gradUy(double x, double y, double t, double T_period){
    double temp;
    double signum = cos(pi * t/T_period);
    if(signum == 0)
        signum = 0;
    else if(signum < 0)
        signum = -1;
    else
        signum = 1;
    temp = 0;
    return temp;
}

__device__ double gradVx(double x, double y, double t, double T_period){
    double temp;
    double signum = cos(pi * t/T_period);
    if(signum == 0)
        signum = 0;
    else if(signum < 0)
        signum = -1;
    else
        signum = 1;
    temp = 0;
    return temp;
}

__device__ double gradVy(double x, double y, double t, double T_period){
    double temp;
    double signum = cos(pi * t/T_period);
    if(signum == 0)
        signum = 0;
    else if(signum < 0)
        signum = -1;
    else
        signum = 1;
    temp = signum;
    return temp;
}
#endif
