#include "hip/hip_runtime.h"
//
//  RayleighVelocityCUDA.cu
//
//
//  Created by Raunak Bardia on 12/18/17.
//
//

#ifndef _RayleighVelocityCUDA_cu
#define _RayleighVelocityCUDA_cu

__device__ __host__ double Velx(double x, double y, double t, double T_period){
    double temp;
    temp = -(1 - exp(c * pow(x,na))) * (U0 * cos(k * y) + U1 * cos(2 * k * y)) * exp(omega * t);
    return temp;
}

__device__ __host__ double Vely(double x, double y,double t, double T_period){
    double temp;
    temp = -(c * na * pow(x, na - 1))/(2 * k) * exp(c * pow(x,na) + omega * t) * (2 * U0 * sin(k * y) + U1 * sin(2 * k * y));
    return temp;
}

__device__ double gradUx(double x, double y, double t, double T_period){
    double temp;
    temp = c * na * pow(x, na - 1) * exp(c * pow(x,na) + omega * t) * (U0 * cos(k * y) + U1 * cos(2 * k * y));
    return temp;
}

__device__ double gradUy(double x, double y, double t, double T_period){
    double temp;
    temp = -k * exp(omega * t) * (exp(c * pow(x,na)) - 1) * sin(k * y) * (U0 + 4 * U1 * cos(k * y));
    return temp;
}

__device__ double gradVx(double x, double y, double t, double T_period){
    double temp;
    temp = -c/k * na * pow(x, na - 2) * (c * na * pow(x,na) + na - 1) * sin(k * y) * exp(c * pow(x, na) + omega * t) * (U0 + U1 * cos(k * y));
    return temp;
}

__device__ double gradVy(double x, double y, double t, double T_period){
    double temp;
    temp = -c * na * pow(x, na - 1) * exp(c * pow(x,na) + omega * t) * (U0 * cos(k * y) + U1 * cos(2 * k * y));
    return temp;
}
#endif
